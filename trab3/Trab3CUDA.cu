/**
*   Programação Concorrente - SSC0143 - 2 Semestre de 2015
*   Prof. Dr. Júlio Cezar Estrella
*   Trabalho 3 - Smoothing de imagem utilizando CUDA
*  
*   Alunos:
*       Thiago Ledur Lima       - 8084214
*       Rodrigo Neves Bernardi  - 8066395   
**/



#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
//#include "Smoothing_CUDA.h"
#include <sys/time.h>

#define FILTERDIM 5
#define FILTERSIZE 25
#define TILEWIDTH 16

typedef unsigned char uchar;

typedef struct IMGstructure {
    char type[3];
    int width;
    int height;
    int maxVal;
    uchar *pixel;
    uchar *r;
    uchar *g;
    uchar *b;
} Image;

// verifica erros
void cudaCheck(hipError_t error) {
    if(error != hipSuccess) {
        fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error));
        exit(1);
    }
}

// aloca memória necessária para uma PPMImage, recebendo suas informações
void allocData(Image *img, char* type, int width, int height, int maxVal) {
    strcpy(img->type, type);
    img->width = width;
    img->height = height;
    img->maxVal = maxVal;

    if(img->type[0] == 'P' && img->type[1] == '2') {
        img->pixel = (uchar *)malloc(width * height * sizeof(uchar));
        img->r = NULL;
        img->g = NULL;
        img->b = NULL;
    }

    else if(img->type[0] == 'P' && img->type[1] == '3') {
        img->pixel = NULL;
        img->r = (uchar *)malloc(width * height * sizeof(uchar));
        img->g = (uchar *)malloc(width * height * sizeof(uchar));
        img->b = (uchar *)malloc(width * height * sizeof(uchar));
    }
}

// libera memória usada por uma Image
void freeData(Image *img) {
    
    if (img->type[0] == 'P' && img->type[1] == '2') {
        free(img->pixel);
    }

    else if(img->type[0] == 'P' && img->type[1] == '3') {
        free(img->r);
        free(img->g);
        free(img->b);    
    }
}

// cria uma estrutura Image, incluindo a leitura do arquivo e a chamada da função para alocar a imagem na memória
void readImage(Image *imgIn, Image *imgOut, char *in) {
    FILE *input;
    int i;
    char type[3];
    int maxVal, width, height;

    input = fopen(in, "r");

    if (input == NULL)
        printf("Can't read image file.\n");
    
    else {
        // le tipo
        rewind(input);
        fscanf(input, "%s", type);

        printf("%s\n", type);

        // compara tipo para ver se é pgm (escala de cinza)
        if(type[0] == 'P' && type[1] == '2') {
            // pula fim da linha
            while (getc(input) != '\n');

            // pula comentario da linha
            while (getc(input) == '#') {
                while (getc(input) != '\n');
            }

            // volta um caracter
            fseek(input, -1, SEEK_CUR);

            // le dimensões da imagem e a escala das cores
            fscanf(input, "%d", &width);
            fscanf(input, "%d", &height);
            fscanf(input, "%d", &maxVal);

            // aloca as matrizes width x height das imagens de entrada e saída na memória
            allocData(imgIn, type, width, height, maxVal);
            allocData(imgOut, type, width, height, maxVal);
            
            // le dados do arquivo
            for(i = 0; i < width * height ; i++) {
                // fscanf(input, "%hhu %hhu %hhu", &(imgIn->data->r[i]), &(imgIn->data->g[i]), &(imgIn->data->b[i]));
                fscanf(input, "%hhu\n", &(imgIn->pixel[i]));        
            }
        }

        // se nao, imagem é ppm (colorida)
        else if(type[0] == 'P' && type[1] == '3') {
            // pula fim da linha
            while (getc(input) != '\n');

            // pula comentario da linha
            while (getc(input) == '#') {
                while (getc(input) != '\n');
            }

            // volta um caracter
            fseek(input, -1, SEEK_CUR);

            // le dimensões da imagem e a escala das cores
            fscanf(input, "%d", &width);
            fscanf(input, "%d", &height);
            fscanf(input, "%d", &maxVal);

            // aloca as matrizes width x height das imagens de entrada e saída na memória
            allocData(imgIn, type, width, height, maxVal);
            allocData(imgOut, type, width, height, maxVal);
            
            // le dados do arquivo
            for(i = 0; i < width * height ; i++) {
                // fscanf(input, "%hhu %hhu %hhu", &(imgIn->data->r[i]), &(imgIn->data->g[i]), &(imgIn->data->b[i]));
                fscanf(input, "%hhu %hhu %hhu\n", &(imgIn->r[i]), &(imgIn->g[i]), &(imgIn->b[i]));        
            }
        }
    }

    fclose(input);
}

// cria um arquivo com o resultado guardado na matriz de imagem de saída
void saveImage(Image *img, char *out) {
    FILE *output;
    int i;

    // escreve header contendo tipo, comentário, dimensões e escala
    output = fopen(out, "w");
    fprintf(output, "%s\n", img->type);
    fprintf(output, "#imagem com smooth\n");
    fprintf(output, "%d %d\n", img->width, img->height);
    fprintf(output, "%d\n", img->maxVal);

    // escreve dados da imagem

    if(img->type[0] == 'P' && img->type[1] == '2') {
        for (i = 0; i < img->height * img->width; i++) {
            //fprintf(output, "%d %d %d\t", img->data[i][j].r, img->data[i][j].g, img->data[i][j].b);
            // fprintf(output, "%hhu %hhu %hhu ", img->data->r[i], img->data->g[i], img->data->b[i]);
            
            if((i >= 2*img->width) && (i < ((img->height*img->width) - 2*img->width)))
                fprintf(output, "%hhu\n", img->pixel[i]);
        }
    }

    else if(img->type[0] == 'P' && img->type[1] == '3') {
        for (i = 0; i < img->height * img->width; i++) {
            //fprintf(output, "%d %d %d\t", img->data[i][j].r, img->data[i][j].g, img->data[i][j].b);
            // fprintf(output, "%hhu %hhu %hhu ", img->data->r[i], img->data->g[i], img->data->b[i]);
        
            if((i >= 2*img->width) && (i < ((img->height*img->width) - 2*img->width)))
                fprintf(output, "%hhu %hhu %hhu\n", img->r[i], img->g[i], img->b[i]);
        }
    }

    fclose(output);
}

__global__ void filter(uchar *in, uchar *out, int H, int W) {
    int i, j, k;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int idx = y * W + x;
    float Pvalue = 0;

    k = FILTERDIM / 2;

    // restringe as bordas da imagem
    if (x >= W-k|| y >= H-k|| x <= k-1 || y <= k-1) return;

    // aplica convolucao
    for(i = 0; i <= FILTERDIM - 1; i++) {
        for(j = 0; j <= FILTERDIM - 1; j++) {
           Pvalue += in[(idx - k + i) + W * (j - k)];
        }
    }
    out[idx] = (uchar)(Pvalue / FILTERSIZE);
    //printf("\nFILTER FINISHED IN : %d %d\n", x, y);
}


int main(int argc, char const *argv[]) {
    Image imgIn, imgOut;

    uchar *inR, *inG, *inB, *inPixel, *outR, *outG, *outB, *outPixel;
    int size;
    char in[20], out[20], tempo[20];
    double result;

    // Tempo total
    struct timeval startTimeCudaTotal, endTimeCudaTotal;
    gettimeofday(&startTimeCudaTotal, NULL);

    strcpy(in, "image.ppm");
    strcpy(out, "out_image.ppm");
    strcpy(tempo, "time.txt");

    // le a imagem de entrada
    readImage(&imgIn, &imgOut, in);

    size = imgIn.width * imgIn.height;

    // execucao do programa para imagens em escala de cinza (PGM)
    if(imgIn.type[0] == 'P' && imgIn.type[1] == '2') {
        // Tempo com a alocacao de memoria
        struct timeval startTimeCudaMem, endTimeCudaMem;
        gettimeofday(&startTimeCudaMem, NULL);
    
        cudaCheck(hipMalloc((void**)&inPixel, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&outPixel, size * sizeof(uchar)));
    
        cudaCheck(hipMemcpy(inPixel, imgIn.pixel, size * sizeof(uchar), hipMemcpyHostToDevice));
    
        // define grid e bloco
        dim3 gridDim(imgIn.width / TILEWIDTH + 1, imgIn.height / TILEWIDTH + 1);
        dim3 blockDim(TILEWIDTH, TILEWIDTH);
    
        // tempo do smoothing
        struct timeval startTimeCuda, endTimeCuda;
        gettimeofday(&startTimeCuda, NULL);
            
        // aplica o filter como funcao global usando grid e bloco
        filter<<<gridDim, blockDim>>>(inPixel, outPixel, imgIn.height, imgIn.width);
    
        gettimeofday(&endTimeCuda, NULL);
    
        cudaCheck(hipMemcpy(imgOut.pixel, outPixel, size * sizeof(uchar), hipMemcpyDeviceToHost));
    
        gettimeofday(&endTimeCudaMem, NULL);

        // Salva imagem filtrada
        saveImage(&imgOut, out);

        // Libera memoria
        hipFree(inPixel);
        hipFree(outPixel);

        freeData(&imgIn);
        freeData(&imgOut);

        gettimeofday(&endTimeCudaTotal, NULL);

        // Calcula o tempo
        result = endTimeCuda.tv_sec - startTimeCuda.tv_sec + (endTimeCuda.tv_usec - startTimeCuda.tv_usec) / 1000000.0;
    }

    // Execucao do programa para imagens coloridas (PPM)
    else if(imgIn.type[0] == 'P' && imgIn.type[1] == '3') {
        // Tempo com a alocacao de memoria
        struct timeval startTimeCudaMem, endTimeCudaMem;
        gettimeofday(&startTimeCudaMem, NULL);
    
        cudaCheck(hipMalloc((void**)&inR, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&inG, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&inB, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&outR, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&outG, size * sizeof(uchar)));
        cudaCheck(hipMalloc((void**)&outB, size * sizeof(uchar)));
    
        cudaCheck(hipMemcpy(inR, imgIn.r, size * sizeof(uchar), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(inG, imgIn.g, size * sizeof(uchar), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(inB, imgIn.b, size * sizeof(uchar), hipMemcpyHostToDevice));
    
        // define grid e bloco
        dim3 gridDim(imgIn.width / TILEWIDTH + 1, imgIn.height / TILEWIDTH + 1);
        dim3 blockDim(TILEWIDTH, TILEWIDTH);
    
        // tempo do smoothing
        struct timeval startTimeCuda, endTimeCuda;
        gettimeofday(&startTimeCuda, NULL);
            
        // aplica o filter como funcao global usando grid e bloco
        filter<<<gridDim, blockDim>>>(inR, outR, imgIn.height, imgIn.width);
        filter<<<gridDim, blockDim>>>(inG, outG, imgIn.height, imgIn.width);
        filter<<<gridDim, blockDim>>>(inB, outB, imgIn.height, imgIn.width);
    
        gettimeofday(&endTimeCuda, NULL);
    
        cudaCheck(hipMemcpy(imgOut.r, outR, size * sizeof(uchar), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(imgOut.g, outG, size * sizeof(uchar), hipMemcpyDeviceToHost));
        cudaCheck(hipMemcpy(imgOut.b, outB, size * sizeof(uchar), hipMemcpyDeviceToHost));
    
        gettimeofday(&endTimeCudaMem, NULL);

        // Salva imagem filtrada
        saveImage(&imgOut, out);

        // Libera memoria
        hipFree(inR);
        hipFree(inB);
        hipFree(inG);
        hipFree(outR);
        hipFree(outG);
        hipFree(outB);

        freeData(&imgIn);
        freeData(&imgOut);

        gettimeofday(&endTimeCudaTotal, NULL);

        // Calcula o tempo
        result = endTimeCuda.tv_sec - startTimeCuda.tv_sec + (endTimeCuda.tv_usec - startTimeCuda.tv_usec) / 1000000.0;
    }

    
    //double resultMem = endTimeCudaMem.tv_sec - startTimeCudaMem.tv_sec + (endTimeCudaMem.tv_usec - startTimeCudaMem.tv_usec) / 1000000.0;
    double resultTotal = endTimeCudaTotal.tv_sec - startTimeCudaTotal.tv_sec + (endTimeCudaTotal.tv_usec - startTimeCudaTotal.tv_usec) / 1000000.0;

    printf("GPU: %lfs\n", result);
    printf("GPU + memoria + in/out - TOTAL : %lfs\n", resultTotal);
    
    FILE *t;
    t = fopen(tempo,"w");
    fprintf(t,"GPU: %lfs\n", result);
    fclose(t);

    return 0;
}